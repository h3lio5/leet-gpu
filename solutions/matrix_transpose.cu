#include<hip/hip_runtime.h>
// #include<cstdio>

__global__ void transpose_naive(float* out, const float* in, int width, int height){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        out[x * height + y] = in[y * width + x]; // in[x, y] -> out[y, x]
    }
}

__global__ void transpose_coalesced_write(float* out, const float* in, int width, int height){
    int y = blockIdx.x * blockDim.x + threadIdx.x;
    int x = blockIdx.y * blockDim.y + threadIdx.y;

    if (x< height && y < width) {
        out[y * height + x] = in[x * width + y];
    }
}

int main() {
}